/*  
 * Copyright (c) 2019, NVIDIA CORPORATION.  All rights reserved.
 * 
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are
 * met:
 *  - Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  - Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  - Neither the name(s) of the copyright holder(s) nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 * "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 * LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR 
 * A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
 * HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
 * SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
 * LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
 * DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
 * THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */  

#include <stdlib.h>
#include <stdio.h>

#include <unordered_map>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiptensor.h>
#include <hip/hip_complex.h>


#define HANDLE_ERROR(x)                                               \
{ const auto err = x;                                                 \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                \
  { printf("Error: %s\n", hiptensorGetErrorString(err)); return err; } \
};

#define HANDLE_CUDA_ERROR(x)                                      \
{ const auto err = x;                                             \
  if( err != hipSuccess )                                        \
  { printf("Error: %s\n", hipGetErrorString(err)); return err; } \
};



struct GPUTimer
{
    GPUTimer() 
    {
        hipEventCreate(&start_);
        hipEventCreate(&stop_);
        hipEventRecord(start_, 0);
    }

    ~GPUTimer() 
    {
        hipEventDestroy(start_);
        hipEventDestroy(stop_);
    }

    void start() 
    {
        hipEventRecord(start_, 0);
    }

    float seconds() 
    {
        hipEventRecord(stop_, 0);
        hipEventSynchronize(stop_);
        float time;
        hipEventElapsedTime(&time, start_, stop_);
        return time * 1e-3;
    }
    private:
    hipEvent_t start_, stop_;
};

int main()
{
    typedef hipFloatComplex floatTypeA;
    typedef hipFloatComplex floatTypeB;
    typedef hipFloatComplex floatTypeC;
    typedef hipFloatComplex floatTypeCompute;

    hipDataType typeA = HIP_C_32F;
    hipDataType typeB = HIP_C_32F;
    hipDataType typeC = HIP_C_32F;
    hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

    floatTypeCompute alpha = make_hipFloatComplex(1.0f, 0.f) ;
    floatTypeCompute beta  = make_hipFloatComplex(0.f, 0.f);

    /**********************
     * Computing: C_{m,u,n,v} = alpha * A_{m,h,k,n} B_{u,k,v,h} + beta * C_{m,u,n,v}
     **********************/

    std::vector<int> modeA{'u', 'm'};
    std::vector<int> modeB{'m', 'h'};
    std::vector<int> modeC{'u', 'h'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    std::unordered_map<int, int64_t> extent;
    extent['m'] = 2;
    extent['n'] = 2;
    extent['u'] = 1 << 30;
    extent['v'] = 2;
    extent['h'] = 2;
    extent['k'] = 2;

    double gflops = (2.0 * extent['m'] * extent['n'] * extent['u'] * extent['v'] * extent['k'] * extent['h']) /1e9;

    std::vector<int64_t> extentC;
    for (auto mode : modeC)
        extentC.push_back(extent[mode]);
    std::vector<int64_t> extentA;
    for (auto mode : modeA)
        extentA.push_back(extent[mode]);
    std::vector<int64_t> extentB;
    for (auto mode : modeB)
        extentB.push_back(extent[mode]);

    /**********************
     * Allocating data
     **********************/

    size_t elementsA = 1;
    for (auto mode : modeA)
        elementsA *= extent[mode];
    size_t elementsB = 1;
    for (auto mode : modeB)
        elementsB *= extent[mode];
    size_t elementsC = 1;
    for (auto mode : modeC)
        elementsC *= extent[mode];

    size_t sizeA = sizeof(floatTypeA) * elementsA;
    size_t sizeB = sizeof(floatTypeB) * elementsB;
    size_t sizeC = sizeof(floatTypeC) * elementsC;
    printf("Total memory: %.2f GiB\n", (sizeA + sizeB + sizeC)/1024./1024./1024);

    void *A_d, *B_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc((void**) &A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc((void**) &C_d, sizeC));

    floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
    floatTypeB *B = (floatTypeB*) malloc(sizeof(floatTypeB) * elementsB);
    floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

    if (A == NULL || B == NULL || C == NULL)
    {
        printf("Error: Host allocation of A or C.\n");
        return -1;
    }

    /*******************
     * Initialize data
     *******************/

    for (int64_t i = 0; i < elementsA; i++){
        // A[i].x = (((float) rand())/RAND_MAX - 0.5)*100;
        // A[i].y = (((float) rand())/RAND_MAX - 0.5)*100;
        A[i].x = (float)i;
        A[i].y = (float)0;
    }
        
    for (int64_t i = 0; i < elementsB; i++){
        // B[i].x = (((float) rand())/RAND_MAX - 0.5)*100;
        // B[i].y = (((float) rand())/RAND_MAX - 0.5)*100;
        B[i].x = (float)i;
        B[i].y = (float)0;
    }
        
    for (int64_t i = 0; i < elementsC; i++){
        // C[i].x = (((float) rand())/RAND_MAX - 0.5)*100;
        // C[i].y = (((float) rand())/RAND_MAX - 0.5)*100;
        C[i].x = (float)0;
        C[i].y = (float)0;
    }
        

    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    /*************************
     * cuTENSOR
     *************************/ 

    hiptensorHandle_t handle;
    HANDLE_ERROR(cutensorInit(&handle));

    /**********************
     * Create Tensor Descriptors
     **********************/

    hiptensorTensorDescriptor_t descA;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descA,
                 nmodeA,
                 extentA.data(),
                 NULL,/*stride*/
                 typeA, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descB;
    HANDLE_ERROR(hiptensorInitTensorDescriptor(&handle,
                 &descB,
                 nmodeB,
                 extentB.data(),
                 NULL,/*stride*/
                 typeB, HIPTENSOR_OP_IDENTITY));

    hiptensorTensorDescriptor_t descC;
    HANDLE_ERROR(hiptensorInitTensorDescriptor( &handle,
                 &descC,
                 nmodeC,
                 extentC.data(),
                 NULL,/*stride*/
                 typeC, HIPTENSOR_OP_IDENTITY));

    /**********************************************
     * Retrieve the memory alignment for each tensor
     **********************************************/ 

     uint32_t alignmentRequirementA;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
                  A_d,
                  &descA,
                  &alignmentRequirementA));

     uint32_t alignmentRequirementB;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
                  B_d,
                  &descB,
                  &alignmentRequirementB));

     uint32_t alignmentRequirementC;
     HANDLE_ERROR(cutensorGetAlignmentRequirement(&handle,
                  C_d,
                  &descC, 
                  &alignmentRequirementC));

    /*******************************
     * Create Contraction Descriptor
     *******************************/

    cutensorContractionDescriptor_t desc;
    HANDLE_ERROR(cutensorInitContractionDescriptor(&handle, 
                 &desc,
                 &descA, modeA.data(), alignmentRequirementA,
                 &descB, modeB.data(), alignmentRequirementB,
                 &descC, modeC.data(), alignmentRequirementC,
                 &descC, modeC.data(), alignmentRequirementC,
                 typeCompute));

    /**************************
    * Set the algorithm to use
    ***************************/

    cutensorContractionFind_t find;
    HANDLE_ERROR(cutensorInitContractionFind( 
                 &handle, &find, 
                 HIPTENSOR_ALGO_DEFAULT));

    /**********************
     * Query workspace
     **********************/

    uint64_t worksize = 0;
    HANDLE_ERROR(cutensorContractionGetWorkspaceSize(&handle,
                 &desc,
                 &find,
                 CUTENSOR_WORKSPACE_RECOMMENDED, &worksize));

    void *work = nullptr;
    if (worksize > 0)
    {
        if (hipSuccess != hipMalloc(&work, worksize))
        {
            work = nullptr;
            worksize = 0;
        }
    } 

    /**************************
     * Create Contraction Plan
     **************************/

    hiptensorContractionPlan_t plan;
    HANDLE_ERROR(cutensorInitContractionPlan(&handle,
                 &plan,
                 &desc,
                 &find,
                 worksize));

    /**********************
     * Run
     **********************/

    double minTimeCUTENSOR = 1e100;
    hiptensorStatus_t err;
    for (int i=0; i < 3; ++i)
    {
        hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice);
        hipDeviceSynchronize();

        // Set up timing
        GPUTimer timer;
        timer.start();

        err = hiptensorContraction(&handle,
                                  &plan,
                                  (void*) &alpha, A_d, B_d,
                                  (void*) &beta,  C_d, C_d, 
                                  work, worksize, 0 /* stream */);

        // Synchronize and measure timing
        auto time = timer.seconds();

        if (err != HIPTENSOR_STATUS_SUCCESS)
        {
            printf("ERROR: %s in line %d\n", hiptensorGetErrorString(err), __LINE__);
        }
        minTimeCUTENSOR = (minTimeCUTENSOR < time) ? minTimeCUTENSOR : time;
    }
    
    /**********************
     * Check The Output
     **********************/
    printf("nmodeA: %d\n", nmodeA);
    
    printf("extentA:");
    for (int64_t i: extentA) {
        printf("%lld\t", i);
    }
    printf("\n");
    
    printf("modeA:");
    for (int i: modeA) {
        printf("%c\t", char(i));
    }
    printf("\n");
    
    
    printf("nmodeB: %d\n", nmodeB);
    
    printf("extentB:");
    for (int64_t i: extentB) {
        printf("%lld\t", i);
    }
    printf("\n");
    
    printf("modeB:");
    for (int i: modeB) {
        printf("%c\t", char(i));
    }
    printf("\n");
    
    
    printf("nmodeC: %d\n", nmodeC);
    
    printf("extentC:");
    for (int64_t i: extentC) {
        printf("%lld\t", i);
    }
    printf("\n");
    
    printf("modeC:");
    for (int i: modeC) {
        printf("%c\t", char(i));
    }
    printf("\n");
    
    
        /***************************/
    
    
    printf("A:");
    for (int64_t i = 0; i < elementsA; i++) {
         printf("%f\t", A[i].x);
         printf("%f\t\t", A[i].y);
    }
    printf("\n");

    printf("B:");
    for (int64_t i = 0; i < elementsB; i++) {
         printf("%f\t", B[i].x);
         printf("%f\t\t", B[i].y);
    }
    printf("\n");
    
    HANDLE_CUDA_ERROR(hipMemcpy(C, C_d, sizeC, hipMemcpyDeviceToHost));
    printf("C:");
    for (int64_t i = 0; i < elementsC; i++) {
         printf("%f\t", C[i].x);
         printf("%f\t\t", C[i].y);
    }
    printf("\n");
    /*************************/

    double transferedBytes = sizeC + sizeA + sizeB;
    transferedBytes += ((float) beta.x != 0.f) ? sizeC : 0;
    transferedBytes /= 1e9;
    printf("cuTensor: %.2f GFLOPs/s %.2f GB/s\n", gflops / minTimeCUTENSOR, transferedBytes/ minTimeCUTENSOR);

    if (A) free(A);
    if (B) free(B);
    if (C) free(C);
    if (A_d) hipFree(A_d);
    if (B_d) hipFree(B_d);
    if (C_d) hipFree(C_d);
    if (work) hipFree(work);

    return 0;
}
